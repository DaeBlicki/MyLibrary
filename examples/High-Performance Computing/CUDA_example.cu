#include "hip/hip_runtime.h"
/**
 * @file: CUDA_example (DPHPC Exam HS23 and CUDA Exercises)
 * @brief: The following CUDA kernel implementing basic matrix and vector
 * arithmetic and operations including dense matrix vector multiplication, 
 * matrix addition and vector addition.
 * @date: 05/02/2025
 */

extern "C" {
#include "CUDA_example.h"
#include <hip/hip_runtime.h>
}

// ============================================
// IMPLEMENTATION OF <CUDA> KERNEL FUNCTIONS
// ============================================

/**
 * @name: MatVecMult_kernel
 * @brief: CUDA kernel function for general matrix-vector multiplication
 * @param A: float*, input matrix of floats with size m x n
 * @param x: float*, input vector of floats with size n
 * @param y: float*, output vector of floats with size m
 * @param m: const int, given row size of A and size of y
 * @param n: const int, given col size of A and size of x
 */
__global__ void MatVecMult_kernel(float* A, float* x, float* y, const int m, const int n)
{
    // calculate the idx (tid) in output vector
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid >= m){
        return;
    }

    // calculate inner product for output vector
    float c = 0.;
    for(int i = 0; i < n; i++){
        c = c + x[i] * A[tid * m + i];
    }    
    y[tid] = c;
}

/**
 * @name: MatAdd_kernel
 * @brief: CUDA kernel function for matrix additions
 * @param A: float*, lhs input matrix of floats with size m x n
 * @param B: float*, rhs input matrix of floats with size m x n
 * @param C: float*, output matrix of floats with size m x n
 * @param m: const int, given row size of the matrices
 * @param n: const int, given col size of the matrices
 */
__global__ void MatAdd_kernel(float* A, float* B, float* C, const int m, const int n)
{
    // calculate tid in output matrix
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid >= m){
        return;
    }

    // perform matrix matrix addition
    C[tid] = A[tid] + B[tid];
}

/**
 * @name: VecAdd_kernel
 * @brief: CUDA kernel function for vector additions
 * @param x: float*, lhs input vector of floats with size m x n
 * @param y: float*, rhs input vector of floats with size m x n
 * @param z: float*, output vector of floats with size m x n
 * @param n: const int, given size of the vectors
 */
__global__ void VecAdd_kernel(float* x, float* y, float* z, const int n)
{
    // calculate tid of the output matrix
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid >= n){
        return;
    }

    // perform vector addition
    z[tid] = x[tid] + y[tid];
}

// =========================================================
// IMPLEMENTATION OF <C/C++> FUNCTIONS USING CUDA KERNELS
// =========================================================

/**
 * @name: MatVecMult
 * @brief: <C/C++> function for general matrix-vector multiplication running CUDA kernel
 * @param A: float*, input matrix of floats with size m x n
 * @param x: float*, input vector of floats with size n
 * @param y: float*, output vector of floats with size m
 * @param m: const int, given row size of A and size of y
 * @param n: const int, given col size of A and size of x
 */
void MatVecMult(float* A, float* x, float* y, const int m, const int n)
{
    // CUDA Setting tool variables
    int threads_perblockm = 32;
    dim3 threadsPerBlockm(threads_perblockm);
    int num_blocksm = (int)ceil((float)m/(float)threads_perblockm);
    dim3 numBlocksm(num_blocksm);

    // initialize size of allocation memory
    size_t A_size = m * n * sizeof(float);
    size_t x_size = n * sizeof(float);
    size_t y_size = m * sizeof(float);

    // initialize CUDA kernel parameter variable
    float *A_cuda, *x_cuda, *y_cuda;

    // allocate CUDA memory
    hipMalloc(&A_cuda, A_size);
    hipMalloc(&x_cuda, A_size);
    hipMalloc(&y_cuda, A_size);

    // Memcpy the input in CUDA
    hipMemcpy(A_cuda, A, A_size, hipMemcpyHostToDevice);
    hipMemcpy(x_cuda, x, x_size, hipMemcpyHostToDevice);

    // run on kernel
    MatVecMult_kernel<<<numBlocksm, threadsPerBlockm>>>(
        (float*) A_cuda, (float*) x_cuda, (float*) y_cuda, m, n
    );

    // synchronize host and device
    hipDeviceSynchronize();

    // Memcpy the output in CUDA
    hipMemcpy(y, y_cuda, y_size, hipMemcpyDeviceToHost);

    // free CUDA memory
    hipFree(A_cuda);
    hipFree(x_cuda);
    hipFree(y_cuda);
}

/**
 * @name: MatAdd_kernel
 * @brief: <C/C++> function for matrix additions running CUDA kernel
 * @param A: float*, lhs input matrix of floats with size m x n
 * @param B: float*, rhs input matrix of floats with size m x n
 * @param C: float*, output matrix of floats with size m x n
 * @param m: const int, given row size of the matrices
 * @param n: const int, given col size of the matrices
 */
void MatAdd(float* A, float* B, float* C, const int m, const int n)
{
    // initialize CUDA tool variable
    int threads_perBlock = 32;
    dim3 threadsPerBlock(threads_perBlock);
    int num_blocks = (int) ceil((float) (m * n) / (float) threads_perBlock);
    dim3 numBlocks(num_blocks);

    // allocate size of allocation memory
    size_t mat_size = m * n * sizeof(float);

    // initialize CUDA kernel variable
    float *A_cuda, *B_cuda, *C_cuda;

    // allocate CUDA memory
    hipMalloc(&A_cuda, mat_size);
    hipMalloc(&B_cuda, mat_size);
    hipMalloc(&C_cuda, mat_size);

    // Memcpy the input in CUDA
    hipMemcpy(A_cuda, A, mat_size, hipMemcpyHostToDevice);
    hipMemcpy(B_cuda, B, mat_size, hipMemcpyHostToDevice);

    // run on kernel
    MatAdd_kernel<<<numBlocks, threadsPerBlock>>>(A_cuda, B_cuda, C_cuda, m, n);

    // synchronize host and device
    hipDeviceSynchronize();

    // Memcpy the output in CUDA
    hipMemcpy(C_cuda, C, mat_size, hipMemcpyDeviceToHost);

    // free CUDA memory
    hipFree(A_cuda);
    hipFree(B_cuda);
    hipFree(C_cuda);
}

/**
 * @name: VecAdd_kernel
 * @brief: <C/C++> function for matrix additions running CUDA kernel
 * @param x: float*, lhs input vector of floats with size n
 * @param y: float*, rhs input vector of floats with size n
 * @param z: float*, output vector of floats with size n
 * @param n: const int, given size of the vectors
 */
void VecAdd(float* x, float* y, float* z, const int n)
{
    // CUDA Setting tool variables
    int threads_perBlock = 32;
    dim3 threadsPerBlock(threads_perBlock);
    int num_blocks = (int) ceil((float) n / (float) threads_perBlock);
    dim3 numBlocks(num_blocks);

    // allocate size of input variables
    size_t vec_size = sizeof(float) * n;

    // initialize CUDA kernel variables
    float *x_cuda, *y_cuda, *z_cuda;
    
    // allocate CUDA memory
    hipMalloc(&x_cuda, vec_size);
    hipMalloc(&y_cuda, vec_size);
    hipMalloc(&z_cuda, vec_size);

    // Memcpy the input in CUDA
    hipMemcpy(x_cuda, x, vec_size, hipMemcpyHostToDevice);
    hipMemcpy(y_cuda, y, vec_size, hipMemcpyHostToDevice);

    // run on kernel
    VecAdd_kernel<<<numBlocks, threadsPerBlock>>>(x_cuda, y_cuda, z_cuda, n);

    // synchronize host and device
    hipDeviceSynchronize();

    // Memcpy the output in CUDA
    hipMemcpy(z_cuda, z, vec_size, hipMemcpyDeviceToHost);

    // free CUDA memory
    hipFree(x_cuda);
    hipFree(y_cuda);
    hipFree(z_cuda);
}